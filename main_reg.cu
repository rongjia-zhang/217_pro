/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* 
 * C code for creating the Q data structure for fast convolution-based 
 * Hessian multiplication for arbitrary k-space trajectories.
 *
 * Inputs:
 * kx - VECTOR of kx values, same length as ky and kz
 * ky - VECTOR of ky values, same length as kx and kz
 * kz - VECTOR of kz values, same length as kx and ky
 * x  - VECTOR of x values, same length as y and z
 * y  - VECTOR of y values, same length as x and z
 * z  - VECTOR of z values, same length as x and y
 * phi - VECTOR of the Fourier transform of the spatial basis 
 *      function, evaluated at [kx, ky, kz].  Same length as kx, ky, and kz.
 *
 * recommended g++ options:
 *  -O3 -lm -ffast-math -funroll-all-loops
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>

#include "parboil.h"
#include "file.h"
#include "computeQ.cc"

#include "kernel_reg.cu"

int 
main (int argc, char *argv[]) {
  int numX, numK; 	/* Number of X and K values */
  int original_numK;	/* Number of K values in input file */
  float *kx, *ky, *kz;		/* K trajectory (3D vectors) */
  float *x, *y, *z;		/* X coordinates (3D vectors) */

  float *x_d, *y_d, *z_d;
  float *Qr_d, *Qi_d;
  float *phiI_d, *phiR_d, *phiMag_d;
  struct kValues* kVals_d;

  float *phiR, *phiI;		/* Phi values (complex) */
  float *phiMag;		/* Magnitude of Phi */
  float *Qr, *Qi;		/* Q signal (complex) */
  struct kValues* kVals;

  struct pb_Parameters* params;
  struct pb_TimerSet timers;

  pb_InitializeTimerSet(&timers);

  /* Read command line */
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
    {

      fprintf(stderr, "Expecting one input filename\n");
      exit(-1);
    }
  
  /* Read in data */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  inputData(params->inpFiles[0],
	    &original_numK, &numX,
	    &kx, &ky, &kz,
	    &x, &y, &z,
	    &phiR, &phiI);

  /* Reduce the number of k-space samples if a number is given
   * on the command line */
  if (argc < 2)
    numK = original_numK;
  else
    {
      int inputK;
      char *end;
      inputK = strtol(argv[1], &end, 10);
      if (end == argv[1])
	{
	  fprintf(stderr, "Expecting an integer parameter\n");
	  exit(-1);
	}

      numK = MIN(inputK, original_numK);
    }

  printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
         numX, original_numK, numK);

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  /* Create CPU data structures */
  createDataStructsCPU(numK, numX, &phiMag, &Qr, &Qi);
  
  // Allocate device variables----------------------------------------------phi
  // variables for calculating magnitude of phi 
  hipMalloc((void**)&phiR_d,sizeof(float)*numK);
  hipMalloc((void**)&phiI_d,sizeof(float)*numK);
  hipMalloc((void**)&phiMag_d,sizeof(float)*numK);
  
  hipDeviceSynchronize();
  
  // Copy host variables to device---------------------------------------------
  // variables for calculating magnitude of phi 
  hipMemcpy(phiR_d,phiR,sizeof(float)*numK,hipMemcpyHostToDevice);
  hipMemcpy(phiI_d,phiI,sizeof(float)*numK,hipMemcpyHostToDevice);
  hipMemcpy(phiMag_d,phiMag,sizeof(float)*numK,hipMemcpyHostToDevice);
  
  hipDeviceSynchronize();
  
  // Launch kernel-------------------------------------------------------------
  basicComputePhiMagGPU(numK, phiR_d, phiI_d, phiMag_d);
  
  // Copu device variables to host----------------------------------------------
  hipMemcpy(phiMag,phiMag_d,sizeof(float)*numK,hipMemcpyDeviceToHost);
  
  // Free memory----------------------------------------------------------------
  hipFree(phiR_d);
  hipFree(phiI_d);
  hipFree(phiMag_d);
  
  //--------------------------------------------------------------------------end
  
  kVals = (struct kValues*)calloc(numK, sizeof (struct kValues));
  int k;
  for (k = 0; k < numK; k++) {
    kVals[k].Kx = kx[k];
    kVals[k].Ky = ky[k];
    kVals[k].Kz = kz[k];
    kVals[k].PhiMag = phiMag[k];
  }
  
  // Allocate Device variables---------------------------------------------------Q
  // variables for calculating Q element
  hipMalloc((void**)&x_d, sizeof(float)*numX);
  hipMalloc((void**)&y_d, sizeof(float)*numX);
  hipMalloc((void**)&z_d, sizeof(float)*numX);
  hipMalloc((void**)&Qr_d, sizeof(float)*numX);
  hipMalloc((void**)&Qi_d, sizeof(float)*numX);
  hipMalloc((void**)&kVals_d, sizeof(struct kValues));
  
  hipDeviceSynchronize();
  
  // Copy host variables to device------------------------------------------------
  // variables for calculating Q element
  hipMemcpy(x_d,x,sizeof(float)*numX,hipMemcpyHostToDevice);
  hipMemcpy(y_d,y,sizeof(float)*numX,hipMemcpyHostToDevice);
  hipMemcpy(z_d,z,sizeof(float)*numX,hipMemcpyHostToDevice);
  hipMemcpy(Qr_d,Qr,sizeof(float)*numX,hipMemcpyHostToDevice);
  hipMemcpy(Qi_d,Qi,sizeof(float)*numX,hipMemcpyHostToDevice);
  hipMemcpy(kVals_d,kVals,sizeof(struct kValues),hipMemcpyHostToDevice);
  
  hipDeviceSynchronize();
  
  // Launch kernel-------------------------------------------------------------
  basicComputeQGPU(numK, numX, kVals_d, x_d, y_d, z_d, Qr_d, Qi_d);
  
  // Copu device variables to host----------------------------------------------
  hipMemcpy(Qr,Qr_d,sizeof(float)*numX,hipMemcpyDeviceToHost);
  hipMemcpy(Qi,Qi_d,sizeof(float)*numX,hipMemcpyDeviceToHost);
  
  // Free memory----------------------------------------------------------------
  hipFree(x_d);
  hipFree(y_d);
  hipFree(z_d);
  hipFree(Qr_d);
  hipFree(Qi_d);
  hipFree(kVals_d);
  
  //--------------------------------------------------------------------------end

  if (params->outFile)
    {
      /* Write Q to file */
      pb_SwitchToTimer(&timers, pb_TimerID_IO);
      outputData(params->outFile, Qr, Qi, numX);
      pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
    }

  free (kx);
  free (ky);
  free (kz);
  free (x);
  free (y);
  free (z);
  free (phiR);
  free (phiI);
  free (phiMag);
  free (kVals);
  free (Qr);
  free (Qi);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);

  return 0;
}
