#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/
#define PIx2 6.2831853071795864769252867665590058f
#include <stdio.h>
#include <stdlib.h>

__global__ void ComputePhiMagGPU(int numK, const float* phiR, const float* phiI, float* phiMag){

    /********************************************************************
     *
     * Compute the magnitude of Fourier Transform at each sample point
     *
     ********************************************************************/
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     
     // INSERT KERNEL CODE HERE
     if(tid<numK){
     float real = phiR[tid];
     float imag = phiI[tid];
     phiMag[tid] = real*real + imag*imag;
    }     
}

void basicComputePhiMagGPU(int numK, float* phiR, float* phiI, float* phiMag){

     // Initialize thread block and kernel grid dimensions
     const unsigned int BLOCK_SIZE = 1024;
     dim3 DimGrid((numK-1)/BLOCK_SIZE + 1,1,1);
     dim3 DimBlock(BLOCK_SIZE,1,1);
     
     // Call the kernel for calculating magnitude of Phi
     ComputePhiMagGPU<<<DimGrid,DimBlock>>>(numK, phiR, phiI, phiMag);
}


__global__ void ComputeQGPU(int numK, int numX, const struct kValues* kVals, const float* x, const float* y, const float* z,float* Qr, float* Qi){

    /********************************************************************
     *
     * Calculate Q at each voxel point
     *
     ********************************************************************/
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     
     // INSERT KERNEL CODE HERE
     if(tid<numX){
     Qr[tid] = 0; Qi[tid] = 0;
     }
     
     // register allocate voxel inputs and outputs
     float reg_x = x[tid]; float reg_y = y[tid]; float reg_z = z[tid];
     float reg_Qr = Qr[tid]; float reg_Qi = Qi[tid];
     
     //loop over all the sample points
     for(int m = 0; m < numK; m++){
     
     // 
     float exp = 2 * PI * (kVals[m].Kx * reg_x + kVals[m].Ky * reg_y + kVals[m].Kz * reg_z);

     reg_Qr += kVals[m].PhiMag * cos(exp); reg_Qi += kVals[m].PhiMag * sin(exp);
    } 
    Qr[tid] = reg_Qr; Qi[tid] = reg_Qi;    
}

void basicComputeQGPU(int numK, int numX, struct kValues* kVals, float* x, float* y, float* z,float* Qr, float* Qi){

     // Initialize thread block and kernel grid dimensions
     const unsigned int BLOCK_SIZE = 1024;
     dim3 DimGrid((numX-1)/BLOCK_SIZE + 1,1,1);
     dim3 DimBlock(BLOCK_SIZE,1,1);
     
     // Call the kernel for calculating Q matrix
     ComputeQGPU<<<DimGrid,DimBlock>>>(numK, numX, kVals, x, y, z, Qr, Qi);     
}
